#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <random>
#include <chrono>
#include "common.h"

using namespace std;

__global__ void simulation(int *actual, int *next, int N, int tam, int *move) {
	int index = threadIdx.x + blockIdx.x * N;

	int i_act = actual[index];
	int i_ant = actual[index-1];
	int i_next =actual[index+1];

	if (index==0){
		i_ant = actual[tam-1];
	}

	if (index== tam-1){
		i_next = actual[0];
	}

	int val=0;
	if (i_act==0){
		if (i_ant==1){
			val=1;
		}
		else{
			val= 0;
		}
	}
	else{
		if (i_next==1){
			val= 1;
		}
		else{
			val= 0;
		}
	}

	next[index] = val;
	if (actual[index]==0 && val==1){
		atomicAdd(move, 1);
	}
}

/*
float cspeed(int *actual, int *next, int tam){
	float speed = 0.0;
	for(int i=0;i<tam;i++){
		if(actual[i]==0 && next[i]==1){
			speed+=1.0;
		}
	}
	return speed;
}
*/

int main(int argc, char *argv[]){
	int *actual, *next;
	//float speed = 0.0;
	int tam = atoi(argv[1]);
	int iterations = atoi(argv[2]);
	int *mov;
	//printf("Tam %d\n", tam);
	//printf("Ite %d\n",iterations);
	int size = tam * sizeof(int);

	int *d_actual, *d_next, *d_mov;

	CHECK(hipMalloc((int **)&d_actual, size));
	CHECK(hipMalloc((int **)&d_next, size));
	CHECK(hipMalloc((int **)&d_mov, sizeof(int)))

	actual = (int *)malloc(size);
	next = (int *)malloc(size);
	mov = (int *)malloc(sizeof(int));

	int value;
	float cars =0.0;

	for (int i=0; i<tam; i++){
		value= rand () % (2);
		actual[i]= value;
		cars+=value;
	}

   	memset(next, 0, size);
   	memset(mov, 0, sizeof(int));
	
   	CHECK(hipMemcpy(d_actual, actual, size, hipMemcpyHostToDevice));
   	CHECK(hipMemcpy(d_next, next, size, hipMemcpyHostToDevice));
   	CHECK(hipMemcpy(d_mov, mov, sizeof(int), hipMemcpyHostToDevice));

	std::chrono::steady_clock::time_point _start(std::chrono::steady_clock::now());
	for(int i=0; i<iterations; i++){


		simulation<<<tam/10,10>>>(d_actual, d_next, 10, tam, d_mov);

		/*if (i==0){
			CHECK(hipMemcpy(actual, d_actual, size, hipMemcpyDeviceToHost));

			for (int j=0; j<tam; j++){
	   			printf("%d  ", actual[j]);
	   		}
	   		printf("\n");
		}*/

	   	//CHECK(hipMemcpy(next,d_next, size, hipMemcpyDeviceToHost));
	   	CHECK(hipMemcpy(d_actual, d_next, size, hipMemcpyDeviceToDevice));
	   	CHECK(hipMemcpy(mov, d_mov, sizeof(int), hipMemcpyDeviceToHost));
		
		/*
		speed=cspeed(actual,next,tam);

		CHECK(hipMemcpy(actual,d_next, size, hipMemcpyDeviceToHost));
		
			
	   	for (int j=0; j<tam; j++){
	   		printf("%d  ", next[j]);
	   	}*/
	   	printf("%f,",*mov/cars);
	   	CHECK(hipMemset(d_mov,0,sizeof(int)));
	}
	std::chrono::steady_clock::time_point _end(std::chrono::steady_clock::now());
	std::cout << std::chrono::duration_cast<std::chrono::duration<double>>(_end - _start).count();
	printf("\n");
	free(actual);
	free(next);
	CHECK(hipFree(d_actual));
	CHECK(hipFree(d_next));

	return 0;
}
